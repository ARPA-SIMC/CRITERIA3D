
#include <hip/hip_runtime.h>
extern "C"
__global__ void kernelTest(const double* a1, const double* a2, double* r)
{
    int ind = blockIdx.x;
    r[ind] = a1[ind] + a2[ind];
}

extern "C"
void kernelLauncher(const double* A1, const double* A2, double* R, const uint64_t N)
{

    double *a1, *a2, *r;
    hipMalloc(&a1, N*sizeof(double));
    hipMalloc(&a2, N*sizeof(double));

    hipMemcpy(a1, A1, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(a2, A2, N*sizeof(double), hipMemcpyHostToDevice);

    hipMalloc(&r, N*sizeof(double));

    kernelTest<<<N, 1>>>(a1, a2, r);

    hipMemcpy(R, r, N*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(a1);
    hipFree(a2);
    hipFree(r);
}
